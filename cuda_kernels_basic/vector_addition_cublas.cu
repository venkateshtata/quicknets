#include <hip/hip_runtime.h>

#include <hipblas.h>
#include <assert.h>
#include <math.h>
#include<iostream>

void vector_init(float *a, int n){
    for(int i=0; i<n; i++){
        a[i] = (float)(rand()%100);
    }
}

int main(){

    int n=1<<2;
    size_t bytes = n * sizeof(float);

    float *h_a, *h_b, *h_c;
    float *d_a, *d_b;

    h_a = (float*)malloc(bytes);
    h_b = (float*)malloc(bytes);
    h_c = (float*)malloc(bytes);

    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);

    vector_init(h_a, n);
    vector_init(h_b, n);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipblasSetVector(n, sizeof(float), h_a, 1, d_a, 1);
    hipblasSetVector(n, sizeof(float), h_b, 1, d_b, 1);

    //Laching Saxpy kernel (single precision a * x + y)
    const float scale = 2.0f;
    hipblasSaxpy(handle, n, &scale, d_a, 1, d_b, 1);

    hipblasGetVector(n, sizeof(float), d_b, 1, h_c, 1);

    hipblasDestroy(handle);

    hipFree(d_a);
    hipFree(d_b);
    free(h_a);
    free(h_b);

    printf("Execution Compelted!\n");

    return 0;




}