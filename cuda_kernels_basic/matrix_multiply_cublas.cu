#include <hip/hip_runtime.h>

#include <hipblas.h>
#include <hiprand/hiprand.h>
#include <stdlib.h>
#include <assert.h>
#include <time.h>
#include <stdio.h>
#include <math.h>

int main(){

    int n = 1 << 10;
    size_t bytes = n * n * sizeof(float);

    float *h_a, *h_b, *h_c;
    float *d_a, *d_b, *d_c;

    h_a = (float*)malloc(bytes);
    h_b = (float*)malloc(bytes);
    h_c = (float*)malloc(bytes);
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    hiprandGenerator_t prng;
    hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);

    hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long)clock());
    
    hiprandGenerateUniform(prng, d_a, n*n);
    hiprandGenerateUniform(prng, d_b, n*n);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    float alpha = 1.0f;
    float beta = 2.0f;

    // Calculate: c = (alpha*a) * b + (beta*c)
    // (m X n) * (n * K) = (m X k)
    // Params: handle, operation, operation, m, n, k, aplha, A, lda, B, ldb, B, ldb, beta, C, ldc 
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, d_a, n, d_b, n, &beta, d_c, n);

    hipMemcpy(h_a, d_a, bytes, hipMemcpyDeviceToHost);
    hipMemcpy(h_b, d_b, bytes, hipMemcpyDeviceToHost);
    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

    printf("Execution Completed");

    return 0;



}