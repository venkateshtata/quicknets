
#include <hip/hip_runtime.h>
#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <functional>
#include <iostream>
#include <vector>

using std::cout;
using std::generate;
using std::vector;

__global__ void matMul(const int *a, const int *b, int *c, int N){

	int row = (blockDim.y * blockIdx.y) + threadIdx.y;
	int col = (blockDim.x * blockIdx.x) + threadIdx.x;

	c[row * N + col] = 0;

	for(int i=0; i<N; i++){
		c[row * N + col] += a[row * N + i] * b[i * N + col];
	}
}


int main(){

	int N = 1 << 10;
	size_t bytes = N * N * sizeof(int);

	vector<int> h_a(N * N);
	vector<int> h_b(N * N);
	vector<int> h_c(N * N);

	generate(h_a.begin(), h_a.end(), []() { return rand() % 100; });
	generate(h_b.begin(), h_a.end(), []() { return rand() % 100; });
	
	int *d_a, *d_b, *d_c;
	
	hipMalloc(&d_a, bytes);
	hipMalloc(&d_b, bytes);
	hipMalloc(&d_c, bytes);

	hipMemcpy(d_a, h_a.data(), bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b.data(), bytes, hipMemcpyHostToDevice);
	
	int THREADS = 32;
	int BLOCKS = N/THREADS;

	dim3 threads(THREADS, THREADS);
	dim3 blocks(BLOCKS, BLOCKS);

	matMul<<<blocks, threads>>>(d_a, d_b, d_c, N);

	hipMemcpy(h_c.data(), d_c, bytes, hipMemcpyDeviceToHost);

	cout << "COMPLETED SUCCESSFULLY\n";

 	// Free memory on device
  	hipFree(d_a);
  	hipFree(d_b);
 	hipFree(d_c);

 	return 0;



}

