#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cassert>
#include <iostream>

using std::cout;

__global__ void vectorAdd(int *a, int *b, int *c, int N) {

  int tid = (blockDim.x * blockIdx.x) + threadIdx.x;

  if (tid < N) {
    c[tid] = a[tid] + b[tid];
  }
}

int main() {

  const int N = 1 << 16;
  size_t bytes = N * sizeof(int);

  int *a, *b, *c;

  hipMallocManaged(&a, bytes);
  hipMallocManaged(&b, bytes);
  hipMallocManaged(&c, bytes);
  
  int id = hipGetDevice(&id);

  hipMemAdvise(a, bytes, hipMemAdviseSetPreferredLocation, hipCpuDeviceId);
  hipMemAdvise(b, bytes, hipMemAdviseSetPreferredLocation, hipCpuDeviceId);
  hipMemPrefetchAsync(c, bytes, id);

  for (int i = 0; i < N; i++) {
    a[i] = rand() % 100;
    b[i] = rand() % 100;
  }

  hipMemAdvise(a, bytes, hipMemAdviseSetReadMostly, id);
  hipMemAdvise(b, bytes, hipMemAdviseSetReadMostly, id);
  hipMemPrefetchAsync(a, bytes, id);
  hipMemPrefetchAsync(b, bytes, id);
  
  int BLOCK_SIZE = 1 << 10;

  int GRID_SIZE = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

  vectorAdd<<<GRID_SIZE, BLOCK_SIZE>>>(a, b, c, N);

  hipDeviceSynchronize();

  hipMemPrefetchAsync(a, bytes, hipCpuDeviceId);
  hipMemPrefetchAsync(b, bytes, hipCpuDeviceId);
  hipMemPrefetchAsync(c, bytes, hipCpuDeviceId);

  for (int i = 0; i < N; i++) {
    assert(c[i] == a[i] + b[i]);
  }

  hipFree(a);
  hipFree(b);
  hipFree(c);

  cout << "COMPLETED SUCCESSFULLY!\n";

  return 0;
}