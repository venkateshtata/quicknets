
#include <hip/hip_runtime.h>
#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <iostream>
#include <vector>

using std::cout;
using std:: generate;
using std::vector;

 const int N = 1 << 10;
 const int SHMEM_SIZE = 1 << 10;

 __global__ void matMul(const int *a, const int *b, int *c){

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ int s_a[SHMEM_SIZE];
    __shared__ int s_b[SHMEM_SIZE];

    int tmp = 0;

    for(int i=0; i<N; i+=blockDim.x){

        s_a[threadIdx.y * blockDim.x + threadIdx.x] = a[row * N + i + threadIdx.x];
        s_b[threadIdx.y * blockDim.x + threadIdx.x] = b[i * N + threadIdx.y * N + col];

        __syncthreads();

        for(int j=0; j<blockDim.x; j++){
            tmp += s_a[threadIdx.y * blockDim.x + j] * s_b[j * blockDim.x + threadIdx.x];
        }

        __syncthreads();

    }

    c[row * N + col] = tmp;

 }

 int main(){

    size_t bytes = N * N * sizeof(int);

    vector<int> h_a(N*N);
    vector<int> h_b(N*N);
    vector<int> h_c(N*N);

    generate(h_a.begin(), h_a.end(), []() { return rand() % 100; });
    generate(h_b.begin(), h_b.end(), []() { return rand() % 100; });
 
    int *d_a, *d_b, *d_c;
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);


    hipMemcpy(d_a, h_a.data(), bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b.data(), bytes, hipMemcpyHostToDevice);

    int THREADS = 32;
    int BLOCKS = N/THREADS;

    dim3 threads(THREADS, THREADS);
    dim3 blocks(BLOCKS, BLOCKS);

    matMul<<<blocks, threads>>>(d_a, d_b, d_c);

    hipMemcpy(h_c.data(), d_c, bytes, hipMemcpyDeviceToHost);

    cout<<"Completed execution\n";

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
    
 }






